#include "hip/hip_runtime.h"
__global__ void oddEvenStep(int *input, int *output, int N, int step) {

	int index = BIdx.x * BDim.x + TIdx.x;

	if (index % 2 == step && index < N -1) {
		if (input[index] > input[index+1]) {
			int temp = input[index];
			input[index] = input[index+1];
			input[index+1] = temp;
		}
	}
	__synchThreads();
	output[index] = input[index];

}

#define N (2048*2048)
#define ODD 1
#define EVEN 0
#define THREADS_PER_BLOCK 512
int main(void) {
	int *a, *sorted_a; // host copies of a, b
	int *d_input, *d_output; // device copies of a, b, c
	int size = N * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_input, size);
	hipMalloc((void **)&d_output, size);

	// Alloc space for host copies of a, sorted_a 
	a = (int *)malloc(size); random_ints(a, N);
	sorted_a = (int *)malloc(size); 

	// Copy inputs to device
	hipMemcpy(d_input, a, size, hipMemcpyHostToDevice);

	int K = (N%2==1)?(N+1) / 2:N / 2;

	for (int iteration = 0; iteration < K ; iteration++) {
		oddEvenStep<<<N/THREADS_PER_BLOCK , THREADS_PER_BLOCK >>> (d_input, d_ouput, N, ODD );

		oddEvenStep<<<N/THREADS_PER_BLOCK , THREADS_PER_BLOCK >>> (d_ouput, d_input, N, EVEN );
		
	}

	// Copy result back to host
	cudeaDeviceSynchronize();
	hipMemcpy(c, d_input, size, hipMemcpyDeviceToHost);
	
	// Cleanup
	free(a); free(sorted_a); 
	hipFree(d_input); hipFree(d_output); 
	return 0;
}